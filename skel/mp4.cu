#include "hip/hip_runtime.h"
// MP Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include    <wb.h>

#define BLOCK_SIZE 512

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

__global__ void total(float * input, float * output, int len) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int inx = bx * blockDim.x + tx;

    __shared__ float DS[BLOCK_SIZE * 2];

    // Loading phase
    if(inx < len) DS[tx] = input[inx];
    else DS[tx] = 0.0;

    // Reduction phase
    for(int stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
        int idx = (tx + 1) * stride * 2 - 1;
        __syncthreads();
        if(idx < 2 * BLOCK_SIZE) DS[idx] += DS[idx - stride];
    }

    // Reverse phase
    for(int stride = BLOCK_SIZE/2; stride > 0; stride /= 2) {
	int idx = (tx + 1) * stride * 2 - 1;
        __syncthreads();
	if(idx + stride < 2 * BLOCK_SIZE) {
            DS[idx + stride] += DS[idx];
        }
    }

    // Using the first thread of the block to save the result
    __syncthreads();
    if(inx < len && tx == 0) output[bx] = DS[BLOCK_SIZE * 2 - 1];
}

int main(int argc, char ** argv) {
    int ii;
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numInputElements; // number of elements in the input list
    int numOutputElements; // number of elements in the output list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numInputElements);

    numOutputElements = numInputElements / (BLOCK_SIZE<<1);
    if (numInputElements % (BLOCK_SIZE<<1)) {
        numOutputElements++;
    }
    hostOutput = (float*) malloc(numOutputElements * sizeof(float));

    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numInputElements);
    wbLog(TRACE, "The number of output elements in the input is ", numOutputElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void**) &deviceInput, numInputElements * sizeof(float));
	hipMalloc((void**) &deviceOutput, numOutputElements * sizeof(float));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
	dim3 DimGrid(numOutputElements, 1, 1);
	dim3 DimBlock(BLOCK_SIZE * 2, 1, 1);
		
    wbTime_start(Compute, "Performing CUDA computation");
    
	total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
    hipDeviceSynchronize();
	wbCheck(hipGetLastError());
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying output memory to the CPU");

    /********************************************************************
     * Reduce output vector on the host
     * NOTE: One could also perform the reduction of the output vector
     * recursively and support any size input. For simplicity, we do not
     * require that for this lab.
     ********************************************************************/
    for (ii = 1; ii < numOutputElements; ii++) {
        hostOutput[0] += hostOutput[ii];
    }

    wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(deviceInput);
	hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, 1);

    free(hostInput);
    free(hostOutput);

    return 0;
}


